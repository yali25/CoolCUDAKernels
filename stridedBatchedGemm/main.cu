#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipblas.h>


using std::cout;
using std::cerr;

__global__
void add_matrices(int nblocks, 
                  int out_index,
	          int nm, 
                  int ms, 
                  const double *in_c_matrices, 
                  double *result_matrix)
{
       // start index in the input array and global tid
       const int start_idx = blockIdx.x * blockDim.x + threadIdx.x;
       
       const int start_matrix = (int)(start_idx / (float)ms);
       // cut off the last threads which lie on the last matrix which is not fully covered by threads
       if(nblocks != start_matrix)
       {
           // fetch the first element
           double my_sum = in_c_matrices[start_idx];
           // calculate the the matrix element the current thread processes
           const int out_element = (start_idx % ms);
           const int nn = nblocks*ms;
           for (int i = ((start_matrix + nblocks)*ms+out_element); i < nm; i += nn)
           {
           	my_sum += in_c_matrices[i];
           }
           atomicAdd(&result_matrix[out_element], my_sum);
      }
}


int main()
{
    hipblasHandle_t cublas_handle;
    hipblasStatus_t  blaserr = hipblasCreate(&cublas_handle);

    if(blaserr != HIPBLAS_STATUS_SUCCESS)
    {
       std::cerr << "Could not init the cublas library .\n";
       return 1;
    }

    // 1. Create two large matrices each with 1e6 rows and 16 columns
    const int rows = 1e6;
    const int cols = 16;
    const int msize = rows*cols;
    double *matrixA = new double[msize];
    double *matrixB = new double[msize];
    std::fill(matrixA, matrixA+msize, 1.0);
    std::fill(matrixB, matrixB+msize, 1.0);

    // 2. Allocate memory on the device for the two matrices and the result matrices
    const int batch_size = 32;
    double *cu_matrixA;
    double *cu_matrixB;
    double *cu_matrixC;
    hipMalloc(&cu_matrixA, msize*sizeof(double));    
    hipMalloc(&cu_matrixB, msize*sizeof(double));    
    hipMalloc(&cu_matrixC, (rows/batch_size)*cols*cols*sizeof(double));    

    // 3. copy the data from the host to the device
    hipMemcpy(cu_matrixA, matrixA, msize*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cu_matrixB, matrixB, msize*sizeof(double), hipMemcpyHostToDevice);

    // 4. call gemmBatched 
    const double alpha = 1.0;
    const double beta  = 0.0;
    hipblasDgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, cols, cols, batch_size,
                              &alpha, cu_matrixA, rows, batch_size,
                                      cu_matrixB, rows, batch_size,
                              &beta,  cu_matrixC, cols, cols, rows/batch_size);

    // 5. add up all small matrices to a result matrix
    int blocks  = 1024;   
    int threads = 64;
    int step_size        = floor(((blocks*threads) / (double)(cols*cols)));
    int out_matrix_index = (step_size)*(cols*cols);
    double *result;
    hipMalloc(&result, cols*cols*sizeof(double));
    hipMemset(result,0,cols*cols*sizeof(double));
    add_matrices << <blocks, threads >> > (step_size, out_matrix_index,
                                          (cols*cols)*rows/batch_size, cols*cols, cu_matrixC, result);

    double *cpu_result_buffer = new double[cols*cols];
    hipMemcpy(cpu_result_buffer, result, cols*cols*sizeof(double), hipMemcpyDeviceToHost);

    // 6. Print the result square matrix 
    for(int i=0;i<cols;i++)
    {
      for(int j=0;j<cols;j++)
      {
        std::cout << cpu_result_buffer[i*cols+j] << " ";
      }
      std::cout << "\n";
    }

    // 9. free all memory ...
     
    return 0;
}
